/*****************************************
Emitting C Generated Code
*******************************************/
#include <string.h>
#include <stdlib.h>
#include "cuda_header.h"
#include <stdio.h>
#include <stdint.h>
#include "cublas_header.h"
#include <stdbool.h>
/**************** Snippet ****************/
void Snippet(int x0) {
  float* x1 = (float*)malloc(5 * sizeof(float));
  int x2 = 0;
  while (x2 != 5) {
    x1[x2] = 1.0;
    x2 = x2 + 1;
  }
  float* x3 = (float*)malloc(0 * sizeof(float));
  CUDA_CALL(hipMalloc(&x3, (size_t)(5 * sizeof(float))));
  CUDA_CALL(hipMemcpy(x3, x1, (size_t)(5 * sizeof(float)), hipMemcpyHostToDevice));
  float* x4 = (float*)malloc(0 * sizeof(float));
  CUDA_CALL(hipMalloc(&x4, (size_t)(5 * sizeof(float))));
  CUDA_CALL(hipMemcpy(x4, x1, (size_t)(5 * sizeof(float)), hipMemcpyHostToDevice));
  float* x5 = (float*)malloc(0 * sizeof(float));
  CUDA_CALL(hipMalloc(&x5, (size_t)sizeof(float)));
  hipblasHandle_t x6;
  CUBLAS_CALL(hipblasCreate(&x6));
  CUBLAS_CALL(hipblasSdot(x6, 10, x3, 1, x4, 1, x5));
  float* x7 = (float*)malloc(5 * sizeof(float));
  CUDA_CALL(hipMemcpy(x7, x5, (size_t)sizeof(float), hipMemcpyDeviceToHost));
  printf("Test DOT:\n");
  printf("%f\n", x7[0]);
  CUDA_CALL(hipFree(x3));
  CUDA_CALL(hipFree(x4));
  CUDA_CALL(hipFree(x5));
  CUBLAS_CALL(hipblasDestroy(x6));
}
/*****************************************
End of C Generated Code
*******************************************/
int main(int argc, char *argv[]) {
  if (argc != 2) {
    printf("usage: %s <arg>\n", argv[0]);
    return 0;
  }
  Snippet(atoi(argv[1]));
  return 0;
}
