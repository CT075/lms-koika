/*****************************************
Emitting C Generated Code
*******************************************/
#include <string.h>
#include <stdlib.h>
#include "cuda_header.h"
#include <stdio.h>
#include <stdint.h>
#include "cublas_header.h"
#include <stdbool.h>
/**************** Snippet ****************/
void Snippet(int x0) {
  float* x1 = (float*)malloc(2 * sizeof(float));
  int x2 = 0;
  while (x2 != 2) {
    x1[x2] = 1.0;
    x2 = x2 + 1;
  }
  float* x3 = (float*)malloc(4 * sizeof(float));
  int x4 = 0;
  while (x4 != 4) {
    x3[x4] = 2.0;
    x4 = x4 + 1;
  }
  float* x5 = (float*)malloc(0 * sizeof(float));
  CUDA_CALL(hipMalloc(&x5, (size_t)(4 * sizeof(float))));
  CUDA_CALL(hipMemcpy(x5, x3, (size_t)(4 * sizeof(float)), hipMemcpyHostToDevice));
  float* x6 = (float*)malloc(0 * sizeof(float));
  CUDA_CALL(hipMalloc(&x6, (size_t)(2 * sizeof(float))));
  CUDA_CALL(hipMemcpy(x6, x1, (size_t)(2 * sizeof(float)), hipMemcpyHostToDevice));
  float* x7 = (float*)malloc(0 * sizeof(float));
  CUDA_CALL(hipMalloc(&x7, (size_t)(2 * sizeof(float))));
  CUDA_CALL(hipMemcpy(x7, x1, (size_t)(2 * sizeof(float)), hipMemcpyHostToDevice));
  float* x8 = (float*)malloc(2 * sizeof(float));
  hipblasHandle_t x9;
  CUBLAS_CALL(hipblasCreate(&x9));
  float x10 = 1.0;
  float x11 = 1.0;
  CUBLAS_CALL(hipblasSgemv(x9, HIPBLAS_OP_N, 2, 2, &x10, x5, 2, x6, 1, &x11, x7, 1));
  CUDA_CALL(hipMemcpy(x8, x7, (size_t)(2 * sizeof(float)), hipMemcpyDeviceToHost));
  printf("Test GEMV:\n");
  int x12 = 0;
  while (x12 != 2) {
    printf("%f, ", x8[x12]);
    x12 = x12 + 1;
  }
  CUDA_CALL(hipFree(x5));
  CUDA_CALL(hipFree(x6));
  CUDA_CALL(hipFree(x7));
  CUBLAS_CALL(hipblasDestroy(x9));
}
/*****************************************
End of C Generated Code
*******************************************/
int main(int argc, char *argv[]) {
  if (argc != 2) {
    printf("usage: %s <arg>\n", argv[0]);
    return 0;
  }
  Snippet(atoi(argv[1]));
  return 0;
}
